// Gregory Paton
// 322:451
// CUDA Mandelbrot


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#define		X_RESN	800       /* x resolution */
#define		Y_RESN	800       /* y resolution */

typedef struct complextype
{
    float real, imag;
} Complex;

__global__
void work(int *id, int tb_x, int tb_y, int gr_x, int gr_y)
{
    int i, j, k;
    Complex z, c;
    float lengthsq, temp;
    const int num_threads = tb_x * tb_y;
    int work_width = X_RESN / num_threads;
    const int bid = blockIdx.x + (blockIdx.y * gr_x);
    const int tid = threadIdx.x + (threadIdx.y * tb_x) + (bid * num_threads);
    id[tid] = tid;
    // if X_RESN is not evenly divisible by num_threads
    // give remainder of work to last thread
    if (tid == num_threads - 1)
        work_width = X_RESN - tid;
    
    /*for(i = tid; i < tid + work_width; i++) {
        for(j = 0; j < Y_RESN; j++) {

            z.real = z.imag = 0.0;
            c.real = ((float) j - 400.0)/200.0;
            c.imag = ((float) i - 400.0)/200.0;
            k = 0;

            do  {                              
                temp = z.real*z.real - z.imag*z.imag + c.real;
                z.imag = 2.0*z.real*z.imag + c.imag;
                z.real = temp;
                lengthsq = z.real*z.real+z.imag*z.imag;
                k++;
            } while (lengthsq < 4.0 && k < 100);
            
            if (k == 100) 
                id[i + (j * Y_RESN)] = 1;
            else
                id[i + (j * Y_RESN)] = 0;
        }
    }*/   
}

int main (int argc, char **argv)
{  
    int tb_x = 16;
    int tb_y = 1;
    int gr_x = 1;
    int gr_y = 1;
    if (argc == 5) {
        tb_x = atoi(argv[1]);
        tb_y = atoi(argv[2]);
        gr_x = atoi(argv[3]);
        gr_y = atoi(argv[4]);    
    }
    else {
        printf("usage: %s THREAD_BLOCK_WIDTH THREAD_BLOCK_HEIGHT GRID_WIDTH GRID_HEIGHT\n", argv[0]);
        return -1;
    }
    int id[X_RESN * Y_RESN];
    int *Id;
    hipMalloc((void**)&Id, X_RESN * Y_RESN); 

    dim3 dimBlock(tb_x, tb_y);
    dim3 dimGrid(gr_x, gr_y);

    work<<<dimGrid, dimBlock>>>(Id, tb_x, tb_y, gr_x, gr_y);
    
    hipMemcpy(id, Id, X_RESN * Y_RESN, hipMemcpyDeviceToHost);
    int i;
    for (i = 0; i < 30; ++i)
        printf("id: %d\n", id[i]);
    hipFree(Id);
	/* Program Finished */
    return 0;
}

